/**********************************************************************
 * DESCRIPTION:
 *   Parallel Concurrent Wave Equation w/ CUDA acceleration
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

// CUDA
#define BLOCK_SIZE 1024

// function declaration
void check_param(void);
void update (void);
void printfinal (void);

// global variable
int nsteps,                 	 /* number of time steps */
    tpoints, 	     		 /* total points along string */
    rcode;                  	 /* generic return code */
float values[MAXPOINTS + 2]; 	 /* values at time t */

// CUDA
float  *Vd;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    // check number of points, number of iterations
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n",
                    MINPOINTS, MAXPOINTS);
    }

    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *    CUDA acceleration code block
 *    - Initialize points on line
 *    - Calculate new values using wave equation and update accordingly
 *********************************************************************/
__global__ void runOnGPU(float *Vd, int tpoints, int nsteps)
{
    // variable declaration
    int i, k;
    float x, fac, tmp;
    float dtime, c, dx, tau, sqtau;
    float value, newVal, oldVal;

    // init_line()
    fac = 2.0 * PI;
    k = 1 + blockIdx.x * BLOCK_SIZE + threadIdx.x;
    tmp = tpoints - 1;
    x = (k - 1) / tmp;
    value = sin(fac * x);
    oldVal = value;

    // do_math() + update()
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    if(k <= tpoints) {

        // propagate for nstpes iterations
        for (i = 1; i <= nsteps; i++) {

            // check boundary
            if ((k == 1) || (k == tpoints))
                newVal = 0.0;
            else
                newVal = (2.0 * value) - oldVal + (sqtau * (-2.0) * value);

            // update oldVal and value
            oldVal = value;
            value = newVal;
        }
        
        // update final value to Vd
        Vd[k] = value;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i % 10 == 0)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    // variable declaration
    int size;
    int blockNum;

    // parse arguments
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param();

    printf("Initializing points on the line...\n");
    printf("Updating all points for all time steps...\n");

    // cuda memory allocation
    size = (tpoints + 1) * sizeof(float);
    hipMalloc((void**) &Vd, size);

    // CUDA
    if (tpoints % BLOCK_SIZE == 0) {
        blockNum = tpoints / BLOCK_SIZE;
    } else {
        blockNum = tpoints / BLOCK_SIZE + 1;
    }
    runOnGPU<<<blockNum, BLOCK_SIZE>>>(Vd, tpoints, nsteps);
    hipMemcpy(values, Vd, size, hipMemcpyDeviceToHost);
    hipFree(Vd);

    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    return 0;
}
